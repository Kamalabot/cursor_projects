
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel definition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Reduce vector size for better visualization
    int n = 10;  // Smaller size for demonstration
    size_t size = n * sizeof(float);

    // Host vectors
    float *h_a, *h_b, *h_c;
    // Device vectors
    float *d_a, *d_b, *d_c;

    printf("1. Allocating host memory...\n");
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize vectors with different values
    printf("\n2. Initializing host vectors:\n");
    printf("Vector A: ");
    for (int i = 0; i < n; i++) {
        h_a[i] = i * 1.0f;  // [0,1,2,3,...]
        printf("%.1f ", h_a[i]);
    }
    printf("\nVector B: ");
    for (int i = 0; i < n; i++) {
        h_b[i] = i * 2.0f;  // [0,2,4,6,...]
        printf("%.1f ", h_b[i]);
    }
    printf("\n");

    printf("\n3. Allocating GPU memory...\n");
    hipError_t error;
    error = hipMalloc(&d_a, size);
    if (error != hipSuccess) {
        printf("Error allocating d_a: %s\n", hipGetErrorString(error));
        return -1;
    }
    error = hipMalloc(&d_b, size);
    if (error != hipSuccess) {
        printf("Error allocating d_b: %s\n", hipGetErrorString(error));
        return -1;
    }
    error = hipMalloc(&d_c, size);
    if (error != hipSuccess) {
        printf("Error allocating d_c: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("\n4. Copying data from CPU to GPU...\n");
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    printf("\n5. Configuring CUDA kernel launch parameters...\n");
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    printf("   - Block size: %d\n", blockSize);
    printf("   - Number of blocks: %d\n", numBlocks);
    printf("   - Total threads: %d\n", blockSize * numBlocks);

    printf("\n6. Launching CUDA kernel...\n");
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
    
    // Check for kernel launch errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(error));
        return -1;
    }

    // Wait for GPU to finish
    hipDeviceSynchronize();

    printf("\n7. Copying results back to CPU...\n");
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results
    printf("\n8. Results:\n");
    printf("Vector A: ");
    for (int i = 0; i < n; i++) {
        printf("%.1f ", h_a[i]);
    }
    printf("\nVector B: ");
    for (int i = 0; i < n; i++) {
        printf("%.1f ", h_b[i]);
    }
    printf("\nResult : ");
    for (int i = 0; i < n; i++) {
        printf("%.1f ", h_c[i]);
    }
    printf("\n");

    // Verify results
    printf("\n9. Verifying results...\n");
    bool correct = true;
    for (int i = 0; i < n; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error at index %d: %.1f + %.1f = %.1f (expected: %.1f)\n",
                   i, h_a[i], h_b[i], h_c[i], h_a[i] + h_b[i]);
            correct = false;
            break;
        }
    }
    if (correct) {
        printf("All calculations are correct!\n");
    }

    printf("\n10. Cleaning up memory...\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    printf("\nProgram completed successfully!\n");
    return 0;
}